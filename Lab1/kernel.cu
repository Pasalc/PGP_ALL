#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define CSC(call)  					\
do {								\
	hipError_t res = call;			\
	if (res != hipSuccess) {		\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);					\
	}								\
} while(0)


__global__ void min_vec(double* arr, double* arr2, int n) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	while (idx < n) {
		arr[idx] = arr[idx]<arr2[idx] ? arr[idx] : arr2[idx];
		idx += offset;
	}
}

int main() {
	int n;
	std::cin >> n;
	double* h_arr	= (double*)malloc(sizeof(double) * n);
	double* h_arr2	= (double*)malloc(sizeof(double) * n);

	for (double* arr_p = h_arr; arr_p < h_arr+n; ++arr_p) {
		std::cin >> (*arr_p);
	}
	for (double* arr_p = h_arr2; arr_p < h_arr2+n; ++arr_p) {
		std::cin >> (*arr_p);
	}

	double* d_arr;
	double* d_arr2;

	CSC(hipMalloc(&d_arr, sizeof(double) * n));
	CSC(hipMemcpy(d_arr, h_arr, sizeof(double) * n, hipMemcpyHostToDevice));
	CSC(hipMalloc(&d_arr2, sizeof(double) * n));
	CSC(hipMemcpy(d_arr2, h_arr2, sizeof(double) * n, hipMemcpyHostToDevice));

	hipEvent_t start, end;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end));
	CSC(hipEventRecord(start));

	min_vec << <256, 256 >> > (d_arr, d_arr2, n);
	CSC(hipGetLastError());

	CSC(hipEventRecord(end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));

	//printf("time = %f\n", t);

	CSC(hipMemcpy(h_arr, d_arr, sizeof(double) * n, hipMemcpyDeviceToHost));
	CSC(hipFree(d_arr));
	CSC(hipFree(d_arr2));

	for (int i = 0; i < n; ++i)
		printf("%f ", h_arr[i]);
	printf("\n");
	free(h_arr);
	free(h_arr2);
	return 0;
}
