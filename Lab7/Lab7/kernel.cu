#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <string>
#include <limits>
#include <unistd.h>
#include <iomanip>
#include <fstream>

#define TIMEOUT_TIME 5

double Unext(double* u, int i, int j, double hxs, double hys, int linesize){
	double hyd=1/(hxs*hxs);//Перепутал их?
	double hxd=1/(hys*hys);
	return ((u[(i+1)*linesize+j]+u[(i-1)*linesize+j])*hxd+(u[i*linesize+j+1]+u[i*linesize+j-1])*hyd) / (2*(hxd+hyd));
}

double Calc_local_eps(double* u,double* uprev, int bxs,int bys){
	int i,j; 
	double maxe=0;
	int linesize=bxs+2;
	for(i=1;i<bys+1;++i){
		for(j=1;j<bxs+1;++j){
			maxe = max(maxe,abs(u[i*linesize+j]-uprev[i*linesize+j]));
		}
	}
	return maxe;
}

int main(int argc, char* argv[]) {
	int pxs,pys;
	int bxs,bys;
	std::string out_str("out.txt");
	int str_len;
	double mineps,leps;
	double lx,ly;
	double ul,ur,uu, ud;
	double uzero;
	
	MPI_Init(&argc, &argv);
	int i,j, wrank, wnprocs; 
	MPI_Comm_size(MPI_COMM_WORLD, &wnprocs);
	MPI_Comm_rank(MPI_COMM_WORLD, &wrank);
	if(wrank==0){
		std::cin>>pxs>>pys;
		std::cin>>bxs>>bys;
		std::cin>>out_str;
		std::cin>>mineps;
		std::cin>>lx>>ly;
		std::cin>>ul>>ur>>uu>>ud;
		std::cin>>uzero;
		str_len=out_str.length();
	}
	MPI_Bcast(&pxs,1,MPI_INT,0,MPI_COMM_WORLD);
	MPI_Bcast(&pys,1,MPI_INT,0,MPI_COMM_WORLD);
	MPI_Bcast(&bxs,1,MPI_INT,0,MPI_COMM_WORLD);
	MPI_Bcast(&bys,1,MPI_INT,0,MPI_COMM_WORLD);
	MPI_Bcast(&str_len,1,MPI_INT,0,MPI_COMM_WORLD);
	out_str.resize(str_len);
	MPI_Bcast((void*)out_str.data(),str_len,MPI_CHAR,0,MPI_COMM_WORLD);
	MPI_Bcast(&mineps,1,MPI_DOUBLE,0,MPI_COMM_WORLD);
	
	MPI_Bcast(&lx,1,MPI_DOUBLE,0,MPI_COMM_WORLD);
	MPI_Bcast(&ly,1,MPI_DOUBLE,0,MPI_COMM_WORLD);
	
	MPI_Bcast(&ul,1,MPI_DOUBLE,0,MPI_COMM_WORLD);
	MPI_Bcast(&ur,1,MPI_DOUBLE,0,MPI_COMM_WORLD);
	MPI_Bcast(&uu,1,MPI_DOUBLE,0,MPI_COMM_WORLD);
	MPI_Bcast(&ud,1,MPI_DOUBLE,0,MPI_COMM_WORLD);
	MPI_Bcast(&uzero,1,MPI_DOUBLE,0,MPI_COMM_WORLD);
	
	if(wrank==1){
		fprintf(stderr,"A %d (%d):pxs:%d pys:%d bxs:%d bys:%d lx:%lf ly:%lf ul:%lf ur:%lf uu:%lf ud:%lf uzero:%lf  mineps:%lf %s\n",wrank, wnprocs , pxs,pys,bxs,bys,lx,ly,ul,ur,uu,ud,uzero,mineps, out_str.c_str());
	}
	int n=(bxs+2)*(bys+2);
	double hxs=lx/(bxs*pxs);
	double hys=ly/(bys*pys);
	int linesize=bxs+2;
	
	int color = wrank<pys*pxs ? 0: MPI_UNDEFINED;
	MPI_Comm my_comm;
	MPI_Comm_split(MPI_COMM_WORLD, color, wrank, &my_comm);
	if(color==0){
		char *buf, *buf_p;
		int bsize;
		double* dataswap = (double*)malloc(sizeof(double) * n);
		double* data = (double*)malloc(sizeof(double) * n);
		int rank, nprocs; 
		MPI_Comm_size(my_comm, &nprocs);
		MPI_Comm_rank(my_comm, &rank);
		MPI_Datatype vertvect;
		MPI_Type_vector(bys, 1, bxs+2, MPI_DOUBLE, &vertvect);
		MPI_Type_commit(&vertvect);
	//Init
		int x=rank%pxs,y=rank/pxs;
		int up,down,left,right;
		right = (x+1== pxs)? -1 : y*pxs + (x + 1) % pxs;
		left = (x-1==-1) ? -1 : y*pxs + (x - 1) % pxs;
		up= (y-1==-1) ? -1 :((y-1)*pxs) + x;
		down = (y+1==pys)? -1: ((y+1)*pxs) + x;
		for(i = 0; i < n; i++)
			data[i] = uzero;
		//Init buf
		{
			int lrs, uds;
			MPI_Pack_size(bys,MPI_DOUBLE,my_comm,&lrs);
			MPI_Pack_size(bxs,MPI_DOUBLE,my_comm,&uds);
			int bufsize = 4*MPI_BSEND_OVERHEAD + lrs + uds;
			buf = (char *)malloc( bufsize );
			MPI_Buffer_attach(buf, bufsize);
		}
		
		//Init margins
		if(down==-1){
			for(i = (bys+1)*(bxs+2); i < (bys+2)*(bxs+2); ++i){
				data[i] = ud; 
				dataswap[i]=ud;
			}
		}
		if(up==-1){
			for(i = 0; i < bxs+2; ++i){
				data[i] = uu; 
				dataswap[i]=uu;
			}
		}
		if(left==-1){
			for(i = 0; i < bys+2; ++i){
				data[i*(bxs+2)] = ul;
				dataswap[i*(bxs+2)] = ul;
			}
		}
		if(right==-1){
			for(i = 0; i < bys+2; ++i){
				data[(bxs+1)+i*(bxs+2)] = ur;
				dataswap[(bxs+1)+i*(bxs+2)] = ur;
			}
		}
		//fprintf(stderr, "%d: %d %d %d %d \n", rank, left , right, up , down);
	//main
		int count=0;
		while(count<100000){
			MPI_Request req[4];
			MPI_Status statuses[4];
			//main
			for(i=1;i<bys+1;++i){
				for(j=1;j<bxs+1;++j){
					dataswap[i*linesize+j]=Unext(data,i,j,hxs,hys,linesize);
				}
			}
			std::swap(data,dataswap);
			count++;
			//calc eps
			leps=Calc_local_eps(data,dataswap,bxs,bys);
			if(rank==1){
				//fprintf(stderr, "leps: %lf\n",leps);
			}
			//fprintf(stderr, "%d,%d: after leps\n", rank,count);
			//reduce eps
			//double temp_eps=leps;
			MPI_Allreduce(MPI_IN_PLACE,&leps,1,MPI_DOUBLE,MPI_MAX,my_comm);
			/*
			if(rank==1){
				fprintf(stderr, "leps: %lf\n",leps);
			}
			if(temp_eps==leps){
				fprintf(stderr, "where: %d\n",rank);
			}
			*/
			if(leps<mineps){
				break;
			}
			//fprintf(stderr, "%d,%d: after allred\n", rank,count);
			//!done
			//recv/send from up to down
			int req_count=0;
			if(left!=-1){
				MPI_Bsend((void*)(data+(bxs+2)+1), 1, vertvect, left, 0, my_comm);
				MPI_Irecv((void*)(data+(bxs+2)), 1, vertvect,left,0,my_comm,&req[req_count]);
				++req_count;
			}
			if(right!=-1){
				MPI_Bsend(data+(bxs+2)*2-2, 1, vertvect, right, 0, my_comm);
				MPI_Irecv(data+(bxs+2)*2-1, 1, vertvect, right, 0, my_comm,&req[req_count]);
				++req_count;
			}
			if(up!=-1){
			
				MPI_Bsend(data+(bxs+2)+1, bxs, MPI_DOUBLE, up, 0, my_comm);
				MPI_Irecv(data+1, bxs, MPI_DOUBLE, up, 0, my_comm,&req[req_count]);
				++req_count;
				}
			if(down!=-1){
				MPI_Bsend(data+(bys)*(bxs+2)+1, bxs, MPI_DOUBLE, down, 0, my_comm);
				MPI_Irecv(data+(bys+1)*(bxs+2)+1, bxs, MPI_DOUBLE, down, 0, my_comm,&req[req_count]);
				++req_count;
			}
			//fprintf(stderr, "%d,%d: wait\n", rank,count);
			MPI_Waitall(req_count, req,statuses);
			//fprintf(stderr, "%d,%d: barrier\n", rank,count);
			//MPI_Barrier(my_comm);//not needed
			//fprintf(stderr, "%d,%d: done\n", rank,count);
		}
		MPI_Buffer_detach( &buf_p, &bsize );
	//Write
		if(rank==0){
			std::fstream file(out_str, std::ios::out);
			file << std::scientific << std::setprecision(6);
			for(int yt=0;yt<pys;++yt){
				for(int ly=0;ly<bys;++ly){
					for(int xt=0;xt<pxs;++xt){
						if(yt!=0||xt!=0){
							MPI_Recv(data+(ly+1)*(bxs+2)+1, bxs, MPI_DOUBLE, yt*pxs+xt, 0, my_comm, MPI_STATUS_IGNORE);
						}
						for(int lx=0;lx<bxs ;++lx){
							file<<data[(ly+1)*(bxs+2)+1+lx]<<" ";
						}
					}
					file<<"\n";
				}
			}
			file.close();
		}
		else{
			for(int line=0;line<bys;++line){
				MPI_Send(data + (line+1) * (bxs+2)+1, bxs, MPI_DOUBLE, 0, 0, my_comm);
			}
		}
		
		MPI_Comm_free(&my_comm);
		free(data);
	}
	
	MPI_Finalize();

	return 0;
}